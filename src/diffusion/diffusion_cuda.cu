#include "hip/hip_runtime.h"
#include "diffusion/diffusion_cuda.h"
#include "common/cuda_util.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace diffusion {

__global__ void diffusion_kernel2d(const REAL *f1, REAL *f2,
                                   int nx, int ny,
                                   REAL ce, REAL cw, REAL cn, REAL cs,
                                   REAL cc) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;  
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int c = i + j * nx;
  int w = (i == 0)        ? c : c - 1;
  int e = (i == nx-1)     ? c : c + 1;
  int s = (j == 0)        ? c : c - nx;
  int n = (j == ny-1)     ? c : c + nx;
  f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
      + cn * f1[n];
  return;
}

__global__ void diffusion_kernel3d(const REAL *f1, REAL *f2,
                                   int nx, int ny, int nz,
                                   REAL ce, REAL cw, REAL cn, REAL cs,
                                   REAL ct, REAL cb, REAL cc) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;  
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int c = i + j * nx;
  int xy = nx * ny;
  for (int k = 0; k < nz; ++k) {
    int w = (i == 0)        ? c : c - 1;
    int e = (i == nx-1)     ? c : c + 1;
    int s = (j == 0)        ? c : c - nx;
    int n = (j == ny-1)     ? c : c + nx;
    int b = (k == 0)        ? c : c - xy;
    int t = (k == nz-1)     ? c : c + xy;
    f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
        + cn * f1[n] + cb * f1[b] + ct * f1[t];
    c += xy;
  }
  return;
}

void DiffusionCUDA::InitializeBenchmark() {
  size_t s = sizeof(REAL) * nx_ * ny_ * nz_;
  FORCE_CHECK_CUDA(hipHostMalloc((void**)&f1_, s));
  Initialize(f1_, nx_, ny_, nz_,
             kx_, ky_, kz_, dx_, dy_, dz_,
             kappa_, 0.0, ndim_);
  FORCE_CHECK_CUDA(hipMalloc((void**)&f1_d_, s));
  FORCE_CHECK_CUDA(hipMalloc((void**)&f2_d_, s));
  FORCE_CHECK_CUDA(hipFuncSetCacheConfig(reinterpret_cast<const void*>(diffusion_kernel2d),
                                          hipFuncCachePreferL1));
  FORCE_CHECK_CUDA(hipFuncSetCacheConfig(reinterpret_cast<const void*>(diffusion_kernel3d),
                                          hipFuncCachePreferL1));
  FORCE_CHECK_CUDA(hipEventCreate(&ev1_));
  FORCE_CHECK_CUDA(hipEventCreate(&ev2_));
}

void DiffusionCUDA::FinalizeBenchmark() {
  assert(f1_);
  FORCE_CHECK_CUDA(hipHostFree(f1_));
  assert(f1_d_);
  FORCE_CHECK_CUDA(hipFree(f1_d_));
  assert(f2_d_);
  FORCE_CHECK_CUDA(hipFree(f2_d_));
}


void DiffusionCUDA::RunKernel(int count) {
  size_t s = sizeof(REAL) * nx_ * ny_ * nz_;  
  FORCE_CHECK_CUDA(hipMemcpy(f1_d_, f1_, s, hipMemcpyHostToDevice));

  dim3 block_dim(block_x_, block_y_, block_z_);
  dim3 grid_dim(nx_ / block_x_, ny_ / block_y_, 1);

  assert(nx_ % block_x_ == 0);
  assert(ny_ % block_y_ == 0);
  assert(nz_ % block_z_ == 0);

  CHECK_CUDA(hipEventRecord(ev1_));
  for (int i = 0; i < count; ++i) {
    if (ndim_ == 2) {
      diffusion_kernel2d<<<grid_dim, block_dim>>>
          (f1_d_, f2_d_, nx_, ny_, ce_, cw_, cn_, cs_, cc_);
    } else if (ndim_ == 3) {
      diffusion_kernel3d<<<grid_dim, block_dim>>>
          (f1_d_, f2_d_, nx_, ny_, nz_, ce_, cw_, cn_, cs_, ct_, cb_, cc_);
    }
    REAL *t = f1_d_;
    f1_d_ = f2_d_;
    f2_d_ = t;
  }
  CHECK_CUDA(hipEventRecord(ev2_));
  FORCE_CHECK_CUDA(hipMemcpy(f1_, f1_d_, s, hipMemcpyDeviceToHost));
  FORCE_CHECK_CUDA(hipDeviceSynchronize());
  return;
}

void DiffusionCUDA::DisplayResult(int count, float time) {
  Baseline::DisplayResult(count, time);
  float time_wo_pci;
  hipEventElapsedTime(&time_wo_pci, ev1_, ev2_);
  time_wo_pci *= 1.0e-03;
  printf("Kernel-only performance:\n");
  printf("Elapsed time : %.3f (s)\n", time_wo_pci);
  printf("FLOPS        : %.3f (GFLOPS)\n",
         GetGFLOPS(count, time_wo_pci));
  printf("Throughput   : %.3f (GB/s)\n",
         GetThroughput(count ,time_wo_pci));
}

}

